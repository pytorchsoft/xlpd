#include "hip/hip_runtime.h"
#include "xlpd.h"
#include <iostream>

__global__ void vector_add_kernel(float *a, float *b, float *c, size_t n)
{
    size_t i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
    {
        c[i] = a[i] + b[i];
    }
}

py::array_t<float> vector_add(py::array_t<float> a, py::array_t<float> b)
{
    // Request buffers
    py::buffer_info a_buf = a.request();
    py::buffer_info b_buf = b.request();

    if (a_buf.size != b_buf.size)
    {
        throw std::runtime_error("Input arrays must have the same size.");
    }

    auto result = py::array_t<float>(a_buf.size);
    py::buffer_info r_buf = result.request();

    // Get pointers
    float *a_h = static_cast<float *>(a_buf.ptr);
    float *b_h = static_cast<float *>(b_buf.ptr);
    float *r_h = static_cast<float *>(r_buf.ptr);

    size_t n = a_buf.size;
    size_t size = a_buf.size * sizeof(float);
    std::cout << "size: " << size << std::endl;
    float *a_d, *b_d, *r_d;
    CHECK(hipMalloc((void **)&a_d, size));
    CHECK(hipMalloc((void **)&b_d, size));
    CHECK(hipMalloc((void **)&r_d, size));

    CHECK(hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice));

    vector_add_kernel<<<ceil(n / 256.0f), 256>>>(a_d, b_d, r_d, n);

    CHECK(hipMemcpy(r_h, r_d, size, hipMemcpyDeviceToHost));

    CHECK(hipFree(a_d));
    CHECK(hipFree(b_d));
    CHECK(hipFree(r_d));

    return result;
}
