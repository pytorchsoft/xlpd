#include "hip/hip_runtime.h"
#include "xlpd.h"

#define BLUR_SIZE 2

__global__ void blur_kernel(unsigned char *in, unsigned char *out, int width, int height)
{

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col >= width || row >= height)
        return;

    int pixVal = 0;
    int pixels = 0;

    // Computer the average of the neighboring pixels
    for (int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE + 1; ++blurrow)
    {
        for (int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE + 1; ++blurcol)
        {
            int currow = row + blurrow;
            int curcol = col + blurcol;

            // Check if the current pixel is in the image
            if (curcol < 0 || curcol >= width || currow < 0 || currow >= height)
                continue;

            pixVal += in[currow * width + curcol];
            ++pixels; // Count the number of pixel values that have been added
        }
    }
    // printf("%d %d\n", pixVal, pixels);
    // Write out the result for this pixel
    out[row * width + col] = (unsigned char)((float)pixVal / pixels);
}

py::array_t<uint8_t> blur(py::array_t<uint8_t> pin)
{
    ssize_t height = pin.shape(0);
    ssize_t width = pin.shape(1);
    auto pout = py::array_t<uint8_t>({height, width});
    uint8_t *pin_h = pin.mutable_data();
    uint8_t *pout_h = pout.mutable_data();
    uint8_t *pin_d;
    uint8_t *pout_d;

    CHECK(hipMalloc((void **)&pin_d, pin.size()));
    CHECK(hipMalloc((void **)&pout_d, pout.size()));
    CHECK(hipMemcpy(pin_d, pin_h, pin.size(), hipMemcpyHostToDevice));

    unsigned int w = ceil(width / 16.0f);
    unsigned int h = ceil(height / 16.0f);
    dim3 block = {16, 16, 1};
    dim3 grid = {w, h, 1};
    blur_kernel<<<grid, block>>>(pin_d, pout_d, width, height);
    CHECK(hipMemcpy(pout_h, pout_d, pout.size(), hipMemcpyDeviceToHost));
    CHECK(hipFree(pin_d));
    CHECK(hipFree(pout_d));
    CHECK(hipDeviceSynchronize());
    return pout;
}
