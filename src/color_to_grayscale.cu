#include "hip/hip_runtime.h"
#include "xlpd.h"
#include <iostream>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>

namespace py = pybind11;

__global__ void color_to_grayscale_kernel(uint8_t *pin, uint8_t *pout, ssize_t width, ssize_t height)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < width && row < height)
    {
        int gray = row * width + col;
        int rgb = gray * 3;
        uint8_t r = pin[rgb];
        uint8_t g = pin[rgb + 1];
        uint8_t b = pin[rgb + 2];
        pout[gray] = 0.21 * r + 0.71 * g + 0.07 * b;
    }
}

py::array_t<uint8_t> color_to_grayscale(py::array_t<uint8_t> pin)
{
    ssize_t height = pin.shape(0);
    ssize_t width = pin.shape(1);
    auto pout = py::array_t<uint8_t>({height, width});
    uint8_t *pin_h = pin.mutable_data();
    uint8_t *pout_h = pout.mutable_data();
    uint8_t *pin_d;
    uint8_t *pout_d;

    CHECK(hipMalloc((void **)&pin_d, pin.size()));
    CHECK(hipMalloc((void **)&pout_d, pout.size()));
    CHECK(hipMemcpy(pin_d, pin_h, pin.size(), hipMemcpyHostToDevice));

    unsigned int w = ceil(width / 16.0f);
    unsigned int h = ceil(height / 16.0f);
    dim3 block = {16, 16, 1};
    dim3 grid = {w, h, 1};
    color_to_grayscale_kernel<<<grid, block>>>(pin_d, pout_d, width, height);
    CHECK(hipMemcpy(pout_h, pout_d, pout.size(), hipMemcpyDeviceToHost));
    CHECK(hipFree(pin_d));
    CHECK(hipFree(pout_d));
    CHECK(hipDeviceSynchronize());
    return pout;
}
