#include "xlpd.h"
#include <iostream>

void check(hipError_t err, const char *const func, const char *const file, const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << hipGetErrorString(err) << std::endl;
        std::cerr << func << " " << file << ":" << line << std::endl;
        std::exit(EXIT_FAILURE);
    }
}
